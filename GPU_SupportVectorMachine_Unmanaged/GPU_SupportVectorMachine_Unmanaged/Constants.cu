
#include <hip/hip_runtime.h>
typedef double svm_precision;
#define thread_group_size 64

struct constantBuffer{
	svm_precision	cb_kernelParam1;
	svm_precision	cb_kernelParam2;
	unsigned int	cb_instanceLength;
	unsigned int	cb_instanceCount;
	unsigned int	cb_classIndex;

	// Run flags
	unsigned int	cb_kernel;
	svm_precision	cb_param1;
	svm_precision	cb_param2;
	int				cb_ind1;
	int				cb_ind2;
};

struct evalStruct{
	int ind1;
	int ind2;
	int ind3;
};