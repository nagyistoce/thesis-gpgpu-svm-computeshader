#include "hip/hip_runtime.h"
#include "Constants.cu"

__global__ void CUDA_global_selfProd(svm_precision* b_selfProd, svm_precision* b_inputData);
__global__ void CUDA_global_test(svm_precision* b_output, svm_precision* b_selfProd, svm_precision* b_inputData, svm_precision* b_class, svm_precision* b_alpha);
__global__ void CUDA_global_errorCacheUpdate(svm_precision* b_output, svm_precision* b_selfProd, svm_precision* b_inputData);
__global__ void CUDA_global_SVMOutput(svm_precision* b_output, svm_precision* b_selfProd, svm_precision* b_inputData, svm_precision* b_class, svm_precision* b_alpha);
__host__ void CUDA_host_updateConstantBuffer(constantBuffer* src);

extern "C" void CUDA_updateConstantBuffer(void* src){
	CUDA_host_updateConstantBuffer((constantBuffer*)src);
}

extern "C" void CUDA_selfProd(unsigned int num_threads,svm_precision* b_selfProd, svm_precision* b_inputData){
	dim3 grid(int(svm_precision(num_threads)/thread_group_size)+1, 1, 1);
    dim3 threads(thread_group_size, 1, 1);

    CUDA_global_selfProd<<< grid, threads >>>(b_selfProd,b_inputData);
}

extern "C" void CUDA_testInstances(unsigned int num_threads, svm_precision* b_output, svm_precision* b_selfProd, svm_precision* b_inputData, svm_precision* b_class, svm_precision* b_alpha){
	dim3 grid(int(svm_precision(num_threads)/thread_group_size)+1, 1, 1);
    dim3 threads(thread_group_size, 1, 1);

	CUDA_global_test<<< grid, threads >>>(b_output,b_selfProd,b_inputData,b_class,b_alpha);
}

extern "C" void CUDA_lagrangeUpdate(unsigned int num_threads, svm_precision* b_output, svm_precision* b_selfProd, svm_precision* b_inputData){
	dim3 grid(int(svm_precision(num_threads)/thread_group_size)+1, 1, 1);
    dim3 threads(thread_group_size, 1, 1);

	CUDA_global_errorCacheUpdate<<< grid, threads >>>(b_output,b_selfProd,b_inputData);
}

extern "C" void CUDA_SVMOutput(unsigned int num_threads, svm_precision* b_output, svm_precision* b_selfProd, svm_precision* b_inputData, svm_precision* b_class, svm_precision* b_alpha){
	dim3 grid(int(svm_precision(num_threads)/thread_group_size)+1, 1, 1);
    dim3 threads(thread_group_size, 1, 1);

	CUDA_global_SVMOutput<<< grid, threads >>>(b_output,b_selfProd,b_inputData,b_class,b_alpha);
}