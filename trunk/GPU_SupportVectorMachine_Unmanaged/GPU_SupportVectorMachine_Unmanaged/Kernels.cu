#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Constants.cu"

__constant__ constantBuffer cb_constants;

//texture<svm_precision,1> t_inputDataTexture;
//texture<svm_precision,1> t_classTexture;
//texture<svm_precision,1> t_alphaTexture;
texture<unsigned int,1> t_trainingIndsTexture;
texture<unsigned int,1> t_testingIndsTexture;
texture<unsigned int,1> t_inputIndsTexture;

//surface<void,1> s_outputSurface;
//surface<void,1> s_selfProdSurface;

__host__ void CUDA_host_updateConstantBuffer(constantBuffer* src){
	hipMemcpyToSymbol(HIP_SYMBOL(cb_constants),src,sizeof(constantBuffer));
}

__device__ svm_precision CUDA_device_dotProd(unsigned int i1,unsigned int i2, svm_precision* b_inputData){
	svm_precision result = 0;

	for(unsigned int i=0; i<cb_constants.cb_instanceLength; i++){
		if(i != cb_constants.cb_classIndex){
			result += b_inputData[(cb_constants.cb_instanceLength*i1)+i]*b_inputData[(cb_constants.cb_instanceLength*i2)+i];
		}
	}

	return result;
}

__device__ svm_precision CUDA_device_pukKernel(svm_precision dp, int i1, int i2, svm_precision* b_selfProd){
	svm_precision squaredDifference = -2.0 * dp + b_selfProd[i1] + b_selfProd[i2];
	svm_precision intermediate = cb_constants.cb_kernelParam1 * sqrt(squaredDifference);
	return (1.0 / powf(1.0 + intermediate * intermediate,cb_constants.cb_kernelParam2));
}

__device__ svm_precision CUDA_device_rbfKernel(svm_precision dp, int i1, int i2, svm_precision* b_selfProd){
	return (cb_constants.cb_kernelParam1 * (2. * dp - b_selfProd[i1] - b_selfProd[i2]));
}

__device__ svm_precision CUDA_device_evaluateKernel(svm_precision dp, int i1, int i2, svm_precision* b_selfProd){
	if(i1 == i2)
		return 1.0;

	if(cb_constants.cb_kernel == 0)
		return CUDA_device_pukKernel(dp,i1,i2,b_selfProd);
	else if(cb_constants.cb_kernel == 1)
		return CUDA_device_rbfKernel(dp,i1,i2,b_selfProd);
	else
		return 1.0;
}

__global__ void CUDA_global_selfProd(svm_precision* b_selfProd, svm_precision* b_inputData){
	int tid = blockIdx.x*thread_group_size+threadIdx.x;
	if(tid < cb_constants.cb_instanceCount)
		b_selfProd[tid] = CUDA_device_dotProd(tid,tid,b_inputData);
}

__shared__ evalStruct sdata[thread_group_size];
__global__ void CUDA_global_SVMOutput(svm_precision* b_output, svm_precision* b_selfProd, svm_precision* b_inputData, svm_precision* b_class, svm_precision* b_alpha){
	int tid = blockIdx.x*thread_group_size+threadIdx.x;

	if(tid >= cb_constants.cb_instanceCount)
		return;

	unsigned int n = cb_constants.cb_instanceLength;
	unsigned int outputInd = tex1Dfetch(t_inputIndsTexture,tid);

	sdata[threadIdx.x].ind1 = tex1Dfetch(t_trainingIndsTexture,cb_constants.cb_ind1) * n;
	sdata[threadIdx.x].ind2 = tex1Dfetch(t_trainingIndsTexture,outputInd) * n;
	
	svm_precision result1 = 0;
	unsigned int i = 0;
	do{
		if(i != cb_constants.cb_classIndex)
			result1 += b_inputData[i+sdata[threadIdx.x].ind1] * b_inputData[i+sdata[threadIdx.x].ind2];
		i++;
	}while (i < n);

	b_output[tid] = CUDA_device_evaluateKernel(result1,sdata[threadIdx.x].ind1/n,sdata[threadIdx.x].ind2/n,b_selfProd);
	//b_output[tid] = CUDA_device_evaluateKernel(result1,sdata[threadIdx.x].ind1/n,sdata[threadIdx.x].ind2/n,b_selfProd) * b_class[outputInd] * b_alpha[outputInd];
}

__global__ void CUDA_global_errorCacheUpdate(svm_precision* b_output, svm_precision* b_selfProd, svm_precision* b_inputData){
	int tid = blockIdx.x*thread_group_size+threadIdx.x;

	if(tid >= cb_constants.cb_instanceCount)
		return;

	unsigned int n = cb_constants.cb_instanceLength;
	unsigned int outputInd = tex1Dfetch(t_inputIndsTexture,tid);

	sdata[threadIdx.x].ind1 = tex1Dfetch(t_trainingIndsTexture,cb_constants.cb_ind1) * n;
	sdata[threadIdx.x].ind2 = tex1Dfetch(t_trainingIndsTexture,cb_constants.cb_ind2) * n;
	sdata[threadIdx.x].ind3 = tex1Dfetch(t_trainingIndsTexture,outputInd) * n;
	
	svm_precision result1 = 0, result2 = 0;
	unsigned int i = 0;
	do{
		if(i != cb_constants.cb_classIndex)
			result1 += b_inputData[i+sdata[threadIdx.x].ind1] * b_inputData[i+sdata[threadIdx.x].ind3];
		i++;
	}while (i < n);

	i = 0;
	do{
		if(i != cb_constants.cb_classIndex)
			result2 += b_inputData[i+sdata[threadIdx.x].ind2] * b_inputData[i+sdata[threadIdx.x].ind3];
		i++;
	}while (i < n);


	b_output[tid*2] = CUDA_device_evaluateKernel(result1,sdata[threadIdx.x].ind1/n,sdata[threadIdx.x].ind3/n,b_selfProd);
	b_output[(tid*2)+1] = CUDA_device_evaluateKernel(result2,sdata[threadIdx.x].ind2/n,sdata[threadIdx.x].ind3/n,b_selfProd);
	//b_output[tid] =	cb_constants.cb_param1 * CUDA_device_evaluateKernel(result1,sdata[threadIdx.x].ind1/n,sdata[threadIdx.x].ind3/n,b_selfProd) + 
	//				cb_constants.cb_param2 * CUDA_device_evaluateKernel(result2,sdata[threadIdx.x].ind2/n,sdata[threadIdx.x].ind3/n,b_selfProd);
}

__shared__ svm_precision sdataTest[thread_group_size];
__global__ void CUDA_global_test(svm_precision* b_output, svm_precision* b_selfProd, svm_precision* b_inputData, svm_precision* b_class, svm_precision* b_alpha){
	int tid = blockIdx.x*thread_group_size+threadIdx.x;
	int groupIndex = threadIdx.x;

	if(tid >= cb_constants.cb_instanceCount)
		return;

	unsigned int sVecInd = tex1Dfetch(t_inputIndsTexture,tid);

	sdata[groupIndex].ind1 = tex1Dfetch(t_testingIndsTexture,cb_constants.cb_ind1) * cb_constants.cb_instanceLength;
	sdata[groupIndex].ind2 = tex1Dfetch(t_trainingIndsTexture,sVecInd) * cb_constants.cb_instanceLength;
	
	svm_precision result1 = 0;
	unsigned int i = 0;
	do{
		if(i != cb_constants.cb_classIndex)
			result1 += b_inputData[i+sdata[groupIndex].ind1] * b_inputData[i+sdata[groupIndex].ind2];
		i++;
	}while (i < cb_constants.cb_instanceLength);

	b_output[tid] = CUDA_device_evaluateKernel(result1,sdata[groupIndex].ind1/cb_constants.cb_instanceLength,sdata[groupIndex].ind2/cb_constants.cb_instanceLength,b_selfProd) * b_class[sVecInd] * b_alpha[sVecInd];
}